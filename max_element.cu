#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"                                                                                                                                             
//#include ""                                                                                                                                 
#include <stdio.h>                                                                                                                                                      
#include<hip/hip_runtime.h>                                                                                                                                                        
#define n 10                                                                                                                                                            
__global__ void add(int *a, int *maxim) {                                                                                                                               
                                                                                                                                                                        
        int i = blockIdx.x;                                                                                                                                             
                if (i < n) {                                                                                                                                            
                for(i=0;i<n;i++){                                                                                                                                       
                if(a[i]>*maxim)                                                                                                                                         
         *maxim=a[i];                                                                                                                                                   
         }                                                                                                                                                              
         }                                                                                                                                                              
}                                                                                                                                                                       
int main(){                                                                                                                                                             
        int a[n];                                                                                                                                                       
        //      int b[n];                                                                                                                                               
        int i;                                                                                                                                                          
        int maxim;                                                                                                                                                      
                                                                                                                                                                        
        int* dev_a;                                                                                                                                                     
        //      int* dev_b;                                                                                                                                             
        int* dev_max;                                                                                                                                                   
        hipMalloc((void**)&dev_max, sizeof(int));                                                                                                                      
        hipMalloc((void**)&dev_a, n * sizeof(int));                                                                                                                    
                                                                                                                                                                        
        printf("\narray elements (1st):\n");                                                                                                                            
        for(i=0;i<n;i++){                                                                                                                                               
                scanf("%d",&a[i]);                                                                                                                                      
        }                                                                                                                                                               
        maxim = a[0];                                                                                                                                                   
                                                                                                                                                                        
        hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);                                                                                                  
        hipMemcpy(dev_max, &maxim, sizeof(int), hipMemcpyHostToDevice);                                                                                               
        //      dim3 grid(n,1);                                                                                                                                         
        add<<<n,1>>>(dev_a,dev_max);                                                                                                                                    
        hipMemcpy(&maxim, dev_max, sizeof(int), hipMemcpyDeviceToHost);                                                                                               
                                                                                                                                                                        
        printf("\n Max is %d\n",maxim);                                                                                                                                 
        hipFree(dev_max);                                                                                                                                              
        hipFree(dev_a);                                                                                                                                                
                                                                                                                                                                        
                                                                                                                                                                        
        hipDeviceReset();                                                                                                                                              
        return 0;                                                                                                                                                       
