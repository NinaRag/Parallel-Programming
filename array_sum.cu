#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"                                                                                                                                             
//#include ""                                                                                                                                 
#include <stdio.h>                                                                                                                                                      
#include<hip/hip_runtime.h>                                                                                                                                                        
#define n 15                                                                                                                                                             
__global__ void sum_array(int *a, int *c) {                                                                                                                           

        int i = threadIdx.x;
        int j;
        for(j=blockDim.x>>1, j>0;j>>=1){
        _syncthreads();
        if (i < j) {
                c = a[i] + a[i+1];     
        }  
        }
}                                                                                                                                                                       
int main(){                                                                                                                                                             
        int a[n];                                                                                                                                                                                                                                                                                                              
        int i;                                                                                                                                                          
        int c;                                                                                                                                                  
                                                                                                                                                                        
        int* dev_a;                                                                                                                                                                                                                                                                                                      
        int* dev_c;                                                                                                                                                     
                                                                                                                            
        hipMalloc((void**)&dev_a, n * sizeof(int));                                                                                                                    
        hipMalloc((void**)&dev_c, sizeof(int));                                                                                                                    
        printf("\narray elements (1st):\n");                                                                                                                            
        for(i=0;i<n;i++){                                                                                                                                               
        scanf("%d",&a[i]);                                                                                                                                              
        }                                                                                                                                                                                                                                                                                                                            
        hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);                                                                                                  
                                                                                                          
        sum_array<<<1,n/2>>>(dev_a, dev_c);                                                                                                                              
        hipMemcpy(dev_b, b, n* sizeof(int), hipMemcpyHostToDevice);                                                                                                    
        printf("\nsum is\n");                                                                                                         
        printf("%d\n",c);                                                                                                                                                                                                                                                                                                 
                                                                                                                                                                        
        hipFree(dev_c);                                                                                                                                                
        hipFree(dev_a);                                                                                                                                                
                                                                                                                                                      
                                                                                                                                                                        
        hipDeviceReset();                                                                                                                                              
        return 0;                                                                                                                                                       
}               
