#include "stdio.h"                                                                                                                                                      
#include <hip/hip_runtime.h>                                                                                                                                                       
#define COLUMNS 2                                                                                                                                                       
#define ROWS 2                                                                                                                                                          
__global__ void add(int *a, int *b, int *c) {                                                                                                                           
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
                                                                                                                                                                        
        for( int k =0;k<COLUMNS;k++){                                                                                                                                   
                                                                                                                                                                        
        int i = (COLUMNS*y) + k;                                                                                                                                        
        int f = (COLUMNS*y) + x;                                                                                                                                        
        int j = (COLUMNS*k) + x;                                                                                                                                        
                                                                                                                                                                        
                                                                                                                                                                        
        c[f] = c[f] + a[i]*b[j]; 
        }                                                                                                                                                               
}                                                                                                                                                                       
int main() {                                                                                                                                                            
        int a[ROWS][COLUMNS], b[ROWS][COLUMNS], c[ROWS][COLUMNS]; 
        int *dev_a, *dev_b, *dev_c; 
        hipMalloc((void **) &dev_a, ROWS*COLUMNS*sizeof(int));  
        hipMalloc((void **) &dev_b, ROWS*COLUMNS*sizeof(int));                                                                                                         
        hipMalloc((void **) &dev_c, ROWS*COLUMNS*sizeof(int));                                                                                                         
        for (int y = 0; y < ROWS; y++)                                                                                                                                  
                for (int x = 0; x < COLUMNS; x++)  {  
                        scanf("%d",&a[y][x]);                                                                                                                           
                }                                                                                                                                                       
        for (int y = 0; y < ROWS; y++)                                                                                                                                  
                for (int x = 0; x < COLUMNS; x++)  {   
                        scanf("%d",&b[y][x]);                 
        }                                                                                                                                                               
        hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int), hipMemcpyHostToDevice);  
        hipMemcpy(dev_b, b, ROWS*COLUMNS*sizeof(int), hipMemcpyHostToDevice);                                                                                         
        dim3 grid(COLUMNS,ROWS);
        add<<<grid,1>>>(dev_a, dev_b, dev_c);  
        hipMemcpy(c, dev_c, ROWS*COLUMNS*sizeof(int), hipMemcpyDeviceToHost);                                                                                         
        for (int y = 0; y < ROWS; y++) {
                for (int x = 0; x < COLUMNS; x++)  {  
                        printf("[%d][%d]=%d ",y,x,c[y][x]);                                                                                                             
                }
                printf("\n"); 
        }
        return 0; 
}